#include "hip/hip_runtime.h"
/**
 * @file chamfer_distance_gpu.cu
 * @brief Calculate the Chamfer Distance between two point clouds using GPU acceleration.
 */

#include <iostream>
#include <vector>
#include <cmath>
#include <string>
#include <limits>
#include <filesystem>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <pcl/io/pcd_io.h>
#include <pcl/point_types.h>
#include <cfloat> // Include this header for FLT_MAX

// CUDA kernel to compute distances between points in two point clouds
__global__ void computeDistances(const float* cloud1_data, int cloud1_size,
                                 const float* cloud2_data, int cloud2_size,
                                 float* distances) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < cloud1_size) {
        float x1 = cloud1_data[3 * idx];
        float y1 = cloud1_data[3 * idx + 1];
        float z1 = cloud1_data[3 * idx + 2];

        float min_dist = FLT_MAX; // Use FLT_MAX here
        for (int i = 0; i < cloud2_size; ++i) {
            float x2 = cloud2_data[3 * i];
            float y2 = cloud2_data[3 * i + 1];
            float z2 = cloud2_data[3 * i + 2];

            float dx = x2 - x1;
            float dy = y2 - y1;
            float dz = z2 - z1;

            float dist = sqrt(dx * dx + dy * dy + dz * dz);
            min_dist = fminf(min_dist, dist);
        }
        distances[idx] = min_dist;
    }
}

// Function to compute Chamfer distance between two point clouds using GPU
void computeChamferDistanceGPU(const std::vector<pcl::PointXYZ>& cloud1,
                                const std::vector<pcl::PointXYZ>& cloud2,
                                float& chamfer_distance1, float& chamfer_distance2) {
    // Convert point clouds to arrays
    std::vector<float> cloud1_data, cloud2_data;
    for (const auto& point : cloud1) {
        cloud1_data.push_back(point.x);
        cloud1_data.push_back(point.y);
        cloud1_data.push_back(point.z);
    }
    for (const auto& point : cloud2) {
        cloud2_data.push_back(point.x);
        cloud2_data.push_back(point.y);
        cloud2_data.push_back(point.z);
    }

    // Allocate GPU memory
    float *d_cloud1, *d_cloud2, *d_distances1, *d_distances2;
    hipMalloc((void **)&d_cloud1, sizeof(float) * cloud1_data.size());
    hipMalloc((void **)&d_cloud2, sizeof(float) * cloud2_data.size());
    hipMalloc((void **)&d_distances1, sizeof(float) * cloud1.size());
    hipMalloc((void **)&d_distances2, sizeof(float) * cloud2.size());

    // Copy data from host to device
    hipMemcpy(d_cloud1, cloud1_data.data(), sizeof(float) * cloud1_data.size(), hipMemcpyHostToDevice);
    hipMemcpy(d_cloud2, cloud2_data.data(), sizeof(float) * cloud2_data.size(), hipMemcpyHostToDevice);

    // Launch CUDA kernel
    int block_size = 256;
    int num_blocks_cloud1 = (cloud1.size() + block_size - 1) / block_size;
    int num_blocks_cloud2 = (cloud2.size() + block_size - 1) / block_size;
    computeDistances<<<num_blocks_cloud1, block_size>>>(d_cloud1, cloud1.size(), d_cloud2, cloud2.size(), d_distances1);
    computeDistances<<<num_blocks_cloud2, block_size>>>(d_cloud2, cloud2.size(), d_cloud1, cloud1.size(), d_distances2);
    hipDeviceSynchronize();

    // Copy data from device to host
    std::vector<float> distances1(cloud1.size()), distances2(cloud2.size());
    hipMemcpy(distances1.data(), d_distances1, sizeof(float) * cloud1.size(), hipMemcpyDeviceToHost);
    hipMemcpy(distances2.data(), d_distances2, sizeof(float) * cloud2.size(), hipMemcpyDeviceToHost);

    // Compute Chamfer distance
    chamfer_distance1 = std::accumulate(distances1.begin(), distances1.end(), 0.0f) / (float)cloud1.size();
    chamfer_distance2 = std::accumulate(distances2.begin(), distances2.end(), 0.0f) / (float)cloud2.size();

    // Free GPU memory
    hipFree(d_cloud1);
    hipFree(d_cloud2);
    hipFree(d_distances1);
    hipFree(d_distances2);
}

int main(int argc, char** argv) {
    // Parse the number of points from command-line argument
    const int num_points = std::stoi(argv[1]);
    // Filenames for input point clouds
    std::string filename1 = "../data_generator/points1_" + std::to_string(num_points) + ".txt";
    std::string filename2 = "../data_generator/points2_" + std::to_string(num_points) + ".txt";
    // Load point cloud 1
    pcl::PointCloud<pcl::PointXYZ>::Ptr cloud1(new pcl::PointCloud<pcl::PointXYZ>);
    cloud1->width = num_points;
    cloud1->height = 1;
    cloud1->points.resize(num_points);

    std::ifstream inFile1(filename1);

    if (!inFile1) {
        std::cerr << "Error: Could not open the file1." << std::endl;
        return 1;
    }

    float x, y, z;
    int count = 0 ;

    std::string line;
    while (inFile1 >> x >> y >> z) {
        cloud1->points[count].x = x;
        cloud1->points[count].y = y;
        cloud1->points[count].z = z;
        count++;
    }
    inFile1.close();

    // Load point cloud 2
    pcl::PointCloud<pcl::PointXYZ>::Ptr cloud2(new pcl::PointCloud<pcl::PointXYZ>);
    cloud2->width = num_points;
    cloud2->height = 1;
    cloud2->points.resize(num_points);

    std::ifstream inFile2(filename2);
    if (!inFile2) {
        std::cerr << "Error: Could not open the file2." << std::endl;
        return 1;
    }

    count = 0 ;
    while (inFile2 >> x >> y >> z) {
        cloud2->points[count].x = x;
        cloud2->points[count].y = y;
        cloud2->points[count].z = z;
        count++;
    }
    inFile2.close();


    float CD = 0.0f;

    // Convert point clouds to vectors
    std::vector<pcl::PointXYZ> points1(cloud1->begin(), cloud1->end());
    std::vector<pcl::PointXYZ> points2(cloud2->begin(), cloud2->end());

    float chamfer_distance1, chamfer_distance2;
    computeChamferDistanceGPU(points1, points2, chamfer_distance1, chamfer_distance2);
    CD = (chamfer_distance1 + chamfer_distance2) /2.0;

    // Output the computed Chamfer Distance
    std::cout << CD << std::endl;

    return 0;
}
